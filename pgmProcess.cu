
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>


/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
  float x1 = p1[1];
  float x2 = p2[1];
  float y1 = p1[0];
  float y2 = p2[0];
  float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));

  return distance;

}

__global__ void dPgmDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius) {
  int ix   = blockIdx.x*blockDim.x + threadIdx.x;
  int iy   = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*numCols + ix;

  int p1[2] = {iy, ix % numCols};
  int p2[2] = {centerRow, centerCol};
  float dis = distance(p1, p2);
  
  if(dis <= radius) {
    pixels[idx] = 0;
  }

}
/*
* Given a slope-intercept form of a line, determines if the point x,y falls on that line.
*
* Note that only the first two values of linearEquation are read.
* the first value is the slope (the m of y=mx+b)
* the second value is the intercept (the b of y=mx+b)
*
*/
__device__ int isOnLine(float *linearEquation, int x, int y ){
    if(ceil((float)x*linearEquation[0] + linearEquation[1]) == y ||
       floor((float)x*linearEquation[0] + linearEquation[1]) == y  ){ //Check floor or ceiling for a thicker line hopefully
        return 1;
    } else {
        return 0;
    }
}
/* 
*
*/
__global__ void dPgmDrawLine(int* pixels, int* p1, int* p2, float* linearEquation, int numRows, int numCols){
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*numCols + ix;

    int left, top, right, bottom;

    //Find boundaries
    if (p1[0] < p2[0]){
        left = p1[0];
        right = p2[0];
    } else {
        left = p2[0];
        right = p1[0];
    }

    if (p1[1] < p2[1]){
        top = p1[1];
        bottom = p2[1];
    } else {
        top = p2[1];
        bottom = p1[1];
    }
    
    

    if (ix < numRows && iy < numCols){ //Check to see if the thread should be interacting with the image

        if (ix >= left && ix <= right && iy >= top && iy <= bottom) { //Check line boundaries
            if (isOnLine(linearEquation,ix,iy)){
                pixels[idx] = 0;
            }
        }
    }
}

