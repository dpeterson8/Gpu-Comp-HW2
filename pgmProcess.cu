
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>


/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
  float x1 = p1[1];
  float x2 = p2[1];
  float y1 = p1[0];
  float y2 = p2[0];
  float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));

  return distance;

}

__device__ float findSlope( int p1[], int p2[]) {
  float x1 = p1[1];
  float x2 = p2[1];
  float y1 = p1[0];
  float y2 = p2[0];
  float slope = ((y2-y1)/(x2-x1));

  return slope;
}

__global__ void dPgmDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius) {
  int ix   = blockIdx.x*blockDim.x + threadIdx.x;
  int iy   = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*numCols + ix;

  int p1[2] = {iy, ix % numCols};
  int p2[2] = {centerRow, centerCol};
  float dis = distance(p1, p2);
  
  if(dis <= radius) {
    pixels[idx] = 0;
  }

}

__global__ void dPgmDrawEdge(int *pixels, int numRows, int numCols, int edgeWidth, char ** header) {
  int ix   = blockIdx.x*blockDim.x + threadIdx.x;
  int iy   = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*numCols + ix;

  if(ix < numCols && iy < numRows) {
    if((ix <= edgeWidth || iy <= edgeWidth) || (ix >= numCols - edgeWidth || iy >= numRows - edgeWidth)) {
      pixels[idx] = 0;
    }
  }
}

__global__ void dPgmDrawLine(int *pixels, int numRows, int numCols, float slope, float rem, int p1row, int p1col) {
  int ix   = blockIdx.x*blockDim.x + threadIdx.x;
  int iy   = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*numCols + ix;

  int p2[2] = {iy, ix % numCols};
  int p1[2] = {p1row, p1col};

  float nSlope = findSlope(p2, p1);
  //float slope2 = slope(newP1, p2);
   //slope1 == slope2
  //iy == ceil(((ix % numCols)*slope) + rem) || iy == floor(((ix % numCols)*slope) + rem)
  if(iy == ((ix % numCols) * slope + rem)) {
    //if(ix > p1row && iy > p1col && ix < p2row && iy < p2col) {
      pixels[idx] = 0;
    //}
  }
}
