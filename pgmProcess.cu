
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>


/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
  float x1 = p1[1];
  float x2 = p2[1];
  float y1 = p1[0];
  float y2 = p2[0];
  float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));

  return distance;

}

__device__ float findSlope( int p1[], int p2[]) {
  float x1 = p1[1];
  float x2 = p2[1];
  float y1 = p1[0];
  float y2 = p2[0];
  float slope = ((y2-y1)/(x2-x1));

  return slope;
}

__global__ void dPgmDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius) {
  int ix   = blockIdx.x*blockDim.x + threadIdx.x;
  int iy   = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*numCols + ix;

  int p1[2] = {iy, ix % numCols};
  int p2[2] = {centerRow, centerCol};
  float dis = distance(p1, p2);
  
  if(dis <= radius) {
    pixels[idx] = 0;
  }

}

__global__ void dPgmDrawEdge(int *pixels, int numRows, int numCols, int edgeWidth, char ** header) {
  int ix   = blockIdx.x*blockDim.x + threadIdx.x;
  int iy   = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*numCols + ix;

  if(ix < numCols && iy < numRows) {
    if((ix <= edgeWidth || iy <= edgeWidth) || (ix >= numCols - edgeWidth || iy >= numRows - edgeWidth)) {
      pixels[idx] = 0;
    }
  }
}

__global__ void dPgmDrawLine(int *pixels, int numCols, float slope, float rem, int p1row, int p1col, int minX, int maxX, int minY, int maxY) {
  int ix   = blockIdx.x*blockDim.x + threadIdx.x;
  int iy   = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*numCols + ix;

  int x = (ix % numCols);
  if(iy == round(((float)x * slope + rem))) {
    if(x >= minX && x <= maxX && iy >= minY && iy <= maxY) {
      pixels[idx] = 0;
    }
  }

}
