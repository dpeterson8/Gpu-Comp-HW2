
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>


/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
    float x1 = p1[1];
    float x2 = p2[1];
    float y1 = p1[0];
    float y2 = p2[0];
    float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));
    
    return distance;

}

__global__ void dPgmDrawCircle(int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header) {
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*numCols + ix;

    int p1[2] = {iy, ix};
    int p2[2] = {centerRow, centerCol};
    float dis = distance(p1, p2);
    //    printf("%f ", dis);

    if((float)radius > dis ) {
        pixels[idx] = 255;
    }

}