
#include <hip/hip_runtime.h>
#include <math.h>

/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
    float x1 = p1[1];
    float x2 = p2[1];
    float y1 = p1[0];
    float y2 = p1[0];
    float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));

    return distance;

}