#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include <sys/time.h>
#include "pgmCpuUtility.h"

int cpuPgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
  
  if (header == NULL) { return 0; }
  if (pixels == NULL) { return 0; }
  
  int i, j;

  for(i = 0; i < numRows; i++) {
    for(j = 0; j < numCols; j++) {

      int p1[2] = {i, j};
      int p2[2] = {centerRow, centerCol};
      int dis = hostDistance(p1, p2);

      if (dis <= radius) {
        pixels[(i * numCols) + j] = 0;
      }
        
    }
  }

  return 1;

}

int cpuPgmDrawEdge( int *pixels, int numRows, int numCols, int edgeWidth, char **header ) {
  
  if (header == NULL) { return 0; }
  if (pixels == NULL) { return 0; }
  
  int i, j;

  for(i = 0; i < numRows; i++) {
    for(j = 0; j < numCols; j++) {
      if(j < numCols && i < numRows) {
        if((j <= edgeWidth || i <= edgeWidth) || (j >= numCols - edgeWidth || i >= numRows - edgeWidth)) {
          pixels[(i * numCols) + j] = 0;
        }
      }
    }
  }

  return 1;

}

int cpuPgmDrawLine( int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ) {
  int i, j;

  float minx, maxx, miny, maxy;
  float slope, intercept;
  float p1r, p1c, p2r, p2c;
  p1r = p1row;
  p1c = p1col;
  p2r = p2row;
  p2c = p2col;

  minx = min(p1col, p2col);
  maxx = max(p1col, p2col);
  miny = min(p1row, p2row);
  maxy = max(p1row, p2row);
  
  if((p2c-p1c) != 0) {
    slope = ((p2r-p1r)/(p2c-p1c));
  } else {
    slope = 0;
  }
  intercept = p2r - (slope * p2c);

  for(i = 0; i < numRows; i++) {
    for(j = 0; j < numCols; j++) {

      if (i == ceil(((float)j * slope) + intercept)) {
        if(j >= minx && j <= maxx && i >= miny && i <= maxy) {
          pixels[(i * numCols) + j] = 0;
        }
      }

    }
  }

  return 1;

}

float hostDistance( int p1[], int p2[] )
{
  float x1 = p1[1];
  float x2 = p2[1];
  float y1 = p1[0];
  float y2 = p2[0];
  float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));
  
  return distance;

}

void displayError() {
  printf("Usage:\n");
  printf("-e edgeWidth oldImageFile newImageFile\n");
  printf("-c circleCenterRow circleCenterCol radius oldImageFile newImageFile\n");
  printf("-l p1row p1col p2row p2col oldImageFile newImageFile");
  printf("You have to run the command with the synopsis: \n\n");

  printf("./programName -e edgeWidth originalImage newImage\n");
  printf("to paint an edge of width edgeWidth in the image of originalIamge\n\n");

  printf("./programName -c circleCenterRow circleCenterCol radius oldImageFile newImageFile\n");
  printf("to paint a big round dot on the image with center at (circleCenterRow,\n");
  printf("circleCenterCol) and radius of radius\n\n");

  printf("./programName -l p1row p1col p2row p2col oldImageFile newImageFile\n");
  printf("to draw a line at a start point with row number = p1row and column\n");
  printf("number = p1col, the line segment ends at a point with row number =\n");
  printf("p2row and column number = p2co\n\n");
}

double currentTime() {
  struct timeval now;
  gettimeofday(&now, NULL);

  return now.tv_sec + now.tv_usec/1000000.0;
}
