#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in) {
    int i, j;

    for( i = 0; i < rowsInHeader; i++) {
        if(header[i] == NULL) {
            return NULL;
        }
        if(fgets(header[i], maxSizeHeadRow, in) == NULL) {
            return NULL;
        }
    }
    
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows);

    int *pixels = (int *) malloc((*numRows * *numCols) * sizeof(int ));

    for(i = 0; i < *numRows; i++) {
        for(j = 0; j < *numCols; j++) {
            if( fscanf(in, "%d ", pixels + ((i * *numCols) + j) ) < 0) {
                return NULL;
            }
        }
    }

    return pixels;
}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
    
    dim3 block, grid;

    block.x = 32;
    block.y = 32;

    grid.x = ceil( (float)numCols / (float)block.x );
    grid.y = ceil( (float)numRows / (float)block.y );

    dPgmDrawCircle<<<grid, block>>>(pixels, numRows, numCols, centerCol, centerRow, radius);
    
    return 1;
}

int cpuPgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
    int i, j;

    for(i = 0; i < numRows; i++) {
        for(j = 0; j < numCols; j++) {
            int p1[2] = {i, j};
            int p2[2] = {centerRow, centerCol};
            int dis = hostDistance(p1, p2);
            if (dis <= radius)
            {
                pixels[(i * numCols) + j] = 0;
            }
            
        }
    }
    return 1;
}

int pgmDrawLine(int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ){

    dim3 block, grid;

    block.x = 32;
    block.y = 32;

    grid.x = ceil( (float)numCols / (float)block.x );
    grid.y = ceil( (float)numRows / (float)block.y );

    int p1[2] = {p1row, p1col};
    int p2[2] = {p2row, p2col};

    float linearEquation[2];
    linearEquation[0] = (float)(p2[1] - p1[1])/(float)(p2[0] - p1[0]);
    linearEquation[1] = (float)p1[1] - (linearEquation[0] * (float)p1[0]);

    dPgmDrawLine<<<grid, block>>>(pixels, p1, p2,linearEquation, numRows, numCols);

    return 1;
}

int cpuPgmDrawLine(int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ){


    return 1;
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out ) {
    
    int i, j;

    for(i = 0; i<rowsInHeader; i++) {
        fprintf(out ,"%s" , *(header + i));
    }

    for(i = 0; i < numRows; i++) {
        for(j = 0; j < numCols; j++) {
            if(j < numCols - 1) {
                fprintf(out, "%d ", *(pixels +((i * numCols) + j)));
            } else {
                fprintf(out, "%d\n", *(pixels +((i * numCols) + j)));
            }
        }
    }
    return 0;
}


float hostDistance( int p1[], int p2[] )
{
    float x1 = p1[1];
    float x2 = p2[1];
    float y1 = p1[0];
    float y2 = p2[0];
    float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));
    
    return distance;
}