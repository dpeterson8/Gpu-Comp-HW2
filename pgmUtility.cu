
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in  ) {
    int i, j;

    for( i = 0; i < rowsInHeader; i++) {
        if(header[i] == NULL) {
            return NULL;
        }
        if(fgets(header[i], maxSizeHeadRow, in) == NULL) {
            return NULL;
        }
    }
    
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows);

    int *pixels = (int *) malloc((*numRows * *numCols) * sizeof(int ));

    for(i = 0; i < *numRows; i++) {
        for(j = 0; j < *numCols; j++) {
            if( fscanf(in, "%d ", pixels + ((i * *numCols) + j) ) < 0) {
                return NULL;
            }
        }
    }

    return pixels;
}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
    

    return 1;
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out ) {
    
    int i, j;

    for(i = 0; i<rowsInHeader; i++) {
        fprintf(out ,"%s" , *(header + i));
    }

    for(i = 0; i < numRows; i++) {
        for(j = 0; j < numCols; j++) {
            if(((i * numCols) + j) % 17 != 0) {
                fprintf(out, "%d ", *(pixels +((i * numCols) + j)));
            } else {
                fprintf(out, "%d\n", *(pixels +((i * numCols) + j)));
            }
        }
    }
    return 0;
}

/*
    temporary main function used in testing the functions used in pgmUtility.cu,
    will be deleted before turnin.
*/

// int main(int argc, char *argv[]) {

//     int i;
//     FILE *in_temp = fopen("balloons.ascii.pgm", "r"); 
//     FILE *out_temp = fopen("balloons.ascii-test.pgm", "w"); 
//     char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));
//     for(i = 0; i < rowsInHeader; i ++) {
//         header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
//     }

//     int numRows, numCols;
//     int * temp = pgmRead(header, &numRows, &numCols, in_temp);
//     int awnser = pgmWrite((const char **) header, temp, numRows, numCols, out_temp);

//     // for(int x = 0; x < 30; x++) {
//     //     printf("%d ", temp[x]);
//     // }
//     dPgmDrawCircle<<<numRows, numCols>>>(temp, numRows, numCols, 0, 0, 0, header);
//     hipDeviceSynchronize();   

//     return 0;
// }