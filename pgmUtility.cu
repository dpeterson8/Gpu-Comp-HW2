#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in  ) {
    int i, j;

    for( i = 0; i < rowsInHeader; i++) {
        if(header[i] == NULL) {
            return NULL;
        }
        if(fgets(header[i], maxSizeHeadRow, in) == NULL) {
            return NULL;
        }
    }
    
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows);

    int *pixels = (int *) malloc((*numRows * *numCols) * sizeof(int ));

    for(i = 0; i < *numRows; i++) {
        for(j = 0; j < *numCols; j++) {
            if( fscanf(in, "%d ", pixels + ((i * *numCols) + j) ) < 0) {
                return NULL;
            }
        }
    }

    return pixels;
}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
    
    dim3 block, grid;

    block.x = 32;
    block.y = 32;

    grid.x = ceil( (float)numCols / (float)block.x );
    grid.y = ceil( (float)numRows / (float)block.y );

    dPgmDrawCircle<<<grid, block>>>(pixels, numRows, numCols, centerCol, centerRow, radius);
    
    return 1;
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out ) {
    
    int i, j;

    for(i = 0; i<rowsInHeader; i++) {
        fprintf(out ,"%s" , *(header + i));
    }

    for(i = 0; i < numRows; i++) {
        for(j = 0; j < numCols; j++) {
            if(j < numCols - 1) {
                fprintf(out, "%d ", *(pixels +((i * numCols) + j)));
            } else {
                fprintf(out, "%d\n", *(pixels +((i * numCols) + j)));
            }
        }
    }
    return 0;
}
