#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in  ) {

  int i, j;

  for( i = 0; i < rowsInHeader; i++) {
    if(header[i] == NULL) {
      return NULL;
    }
    if(fgets(header[i], maxSizeHeadRow, in) == NULL) {
      return NULL;
    }
  }
  
  sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows);

  int *pixels = (int *) malloc((*numRows * *numCols) * sizeof(int ));

  for(i = 0; i < *numRows; i++) {
      for(j = 0; j < *numCols; j++) {
        if( fscanf(in, "%d ", pixels + ((i * *numCols) + j) ) < 0) {
        return NULL;
      }
    }
  }

  return pixels;
}

int pgmDrawEdge( int *pixels, int numRows, int numCols, int edgeWidth, char **header ) {

  if (pixels == NULL) { return 0; }
  if (header == NULL) { return 0; }

  dim3 block, grid;

  block.x = 32;
  block.y = 32;

  grid.x = ceil( (float)numCols / (float)block.x );
  grid.y = ceil( (float)numRows / (float)block.y );

  dPgmDrawEdge<<<grid, block>>>(pixels, numRows, numCols, edgeWidth, header);

  return 1;

}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {

  if (pixels == NULL) { return 0; }
  if (header == NULL) { return 0; }

  if (header == NULL) {
    return 0;
  } else if (pixels == NULL) {
    return 0;
  }
    
  dim3 block, grid;

  block.x = 32;
  block.y = 32;

  grid.x = ceil( (float)numCols / (float)block.x );
  grid.y = ceil( (float)numRows / (float)block.y );

  dPgmDrawCircle<<<grid, block>>>(pixels, numRows, numCols, centerCol, centerRow, radius);
  
  return 1;

}

int pgmDrawLine( int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ) {

  if (pixels == NULL) { return 0; }
  if (header == NULL) { return 0; }
  
  dim3 block, grid;

  block.x = 32;
  block.y = 32;

  int minX, maxX, minY, maxY;
  float p1r, p1c, p2r, p2c;

  p1r = p1row;
  p1c = p1col;
  p2r = p2row;
  p2c = p2col;

  minX = min(p1col, p2col);
  maxX = max(p1col, p2col);
  minY = min(p1row, p2row);
  maxY = max(p1row, p2row);

  grid.x = ceil( (float)numCols / (float)block.x );
  grid.y = ceil( (float)numRows / (float)block.y );

  float slope = ((p2r - p1r)/(p2c - p1c));
  float remainder = p1r - (slope * p1c);

  dPgmDrawLine<<<grid, block>>>(pixels, numCols, slope, remainder, p1row, p1col, minX, maxX, minY, maxY);
  
  return 1;
  
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out ) {

  if (pixels == NULL) { return 0; }
  if (header == NULL) { return 0; }
  
  int i, j;

  for(i = 0; i<rowsInHeader; i++) {
    fprintf(out ,"%s" , *(header + i));
  }

  for(i = 0; i < numRows; i++) {
    for(j = 0; j < numCols; j++) {
      if(j < numCols - 1) {
        fprintf(out, "%d ", *(pixels +((i * numCols) + j)));
      } else {
        fprintf(out, "%d\n", *(pixels +((i * numCols) + j)));
      }
    }
  }
  return 0;
}
