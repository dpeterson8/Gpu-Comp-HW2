#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in  ) {
    int i, j;

    for( i = 0; i < rowsInHeader; i++) {
        if(header[i] == NULL) {
            return NULL;
        }
        if(fgets(header[i], maxSizeHeadRow, in) == NULL) {
            return NULL;
        }
    }
    
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows);

    int *pixels = (int *) malloc((*numRows * *numCols) * sizeof(int ));

    for(i = 0; i < *numRows; i++) {
        for(j = 0; j < *numCols; j++) {
            if( fscanf(in, "%d ", pixels + ((i * *numCols) + j) ) < 0) {
                return NULL;
            }
        }
    }

    return pixels;
}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
    
    dim3 block, grid;

    block.x = 32;
    block.y = 32;

    grid.x = ceil( (float)numCols / (float)block.x );
    grid.y = ceil( (float)numRows / (float)block.y );

    dPgmDrawCircle<<<grid, block>>>(pixels, numRows, numCols, centerCol, centerRow, radius);
    
    return 1;
}

int cpuPgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
    int i, j;

    for(i = 0; i < numRows; i++) {
        for(j = 0; j < numCols; j++) {
            printf("Debug rows: %d cols: %d", numRows, numCols);
            int p1[2] = {i, j};
            int p2[2] = {centerRow, centerCol};
            int dis = hostDistance(p1, p2);
            if (dis <= radius)
            {
                pixels[(i * numCols) + j] = 0;
            }
            
        }
    }
    return 1;
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out ) {
    
    int i, j;

    for(i = 0; i<rowsInHeader; i++) {
        fprintf(out ,"%s" , *(header + i));
    }

    for(i = 0; i < numRows; i++) {
        for(j = 0; j < numCols; j++) {
            if(j < numCols - 1) {
                fprintf(out, "%d ", *(pixels +((i * numCols) + j)));
            } else {
                fprintf(out, "%d\n", *(pixels +((i * numCols) + j)));
            }
        }
    }
    return 0;
}


float hostDistance( int p1[], int p2[] )
{
    float x1 = p1[1];
    float x2 = p2[1];
    float y1 = p1[0];
    float y2 = p2[0];
    float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));
    
    return distance;

}