#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in  ) {

  int i, j;

  for( i = 0; i < rowsInHeader; i++) {
    if(header[i] == NULL) {
      return NULL;
    }
    if(fgets(header[i], maxSizeHeadRow, in) == NULL) {
      return NULL;
    }
  }
  
  sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows);

  int *pixels = (int *) malloc((*numRows * *numCols) * sizeof(int ));

  for(i = 0; i < *numRows; i++) {
      for(j = 0; j < *numCols; j++) {
        if( fscanf(in, "%d ", pixels + ((i * *numCols) + j) ) < 0) {
        return NULL;
      }
    }
  }

  return pixels;
}

int pgmDrawEdge( int *pixels, int numRows, int numCols, int edgeWidth, char **header ) {

  dim3 block, grid;

  block.x = 32;
  block.y = 32;

  grid.x = ceil( (float)numCols / (float)block.x );
  grid.y = ceil( (float)numRows / (float)block.y );

  dPgmDrawEdge<<<grid, block>>>(pixels, numRows, numCols, edgeWidth, header);
  return 1;
}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {

  if (header == NULL) {
    return 0;
  } else if (pixels == NULL) {
    return 0;
  }
    
  dim3 block, grid;

  block.x = 32;
  block.y = 32;

  grid.x = ceil( (float)numCols / (float)block.x );
  grid.y = ceil( (float)numRows / (float)block.y );

  dPgmDrawCircle<<<grid, block>>>(pixels, numRows, numCols, centerCol, centerRow, radius);
  
  return 1;

}

int cpuPgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header ) {
  
  if (header == NULL) {
    return 0;
  } else if (pixels == NULL) {
    return 0;
  }
  
  int i, j;

  for(i = 0; i < numRows; i++) {
    for(j = 0; j < numCols; j++) {

      int p1[2] = {i, j};
      int p2[2] = {centerRow, centerCol};
      int dis = hostDistance(p1, p2);

      if (dis <= radius) {
        pixels[(i * numCols) + j] = 0;
      }
        
    }
  }

  return 1;

}

int pgmDrawLine( int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ) {
  dim3 block, grid;

  block.x = 32;
  block.y = 32;

  grid.x = ceil( (float)numCols / (float)block.x );
  grid.y = ceil( (float)numRows / (float)block.y );

  float slope = ((p2row - p1row)/(p2col - p1col));
  float remainder = p1row - (slope * p1col);

  // if(p1row == (p1col * slope) + remainder) {
  //   printf("Working");
  // }

  dPgmDrawLine<<<grid, block>>>(pixels, numRows, numCols, slope, remainder, p1row, p1col);
  
  return 1;
  
}

int cpuPgmDrawLine( int *pixels, int numRows, int numCols, char **header, int p1row, int p1col, int p2row, int p2col ) {
  int i, j;

  float slope, intercept;
  float p1r, p1c, p2r, p2c;
  p1r = p1row;
  p1c = p1col;
  p2r = p2row;
  p2c = p2col;
  
  slope = ((p2r-p1r)/(p2c-p1c));
  intercept = p2r - (slope * p2c);

  for(i = 0; i < numRows; i++) {
    for(j = 0; j < numCols; j++) {

      if (i == ceil(((float)j * slope) + intercept)) {
        pixels[(i * numCols) + j] = 0;
      }

    }
  }

  return 1;

}


int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out ) {
  
  int i, j;

  for(i = 0; i<rowsInHeader; i++) {
    fprintf(out ,"%s" , *(header + i));
  }

  for(i = 0; i < numRows; i++) {
    for(j = 0; j < numCols; j++) {
      if(j < numCols - 1) {
        fprintf(out, "%d ", *(pixels +((i * numCols) + j)));
      } else {
        fprintf(out, "%d\n", *(pixels +((i * numCols) + j)));
      }
    }
  }
  return 0;
}


float hostDistance( int p1[], int p2[] )
{
  float x1 = p1[1];
  float x2 = p2[1];
  float y1 = p1[0];
  float y2 = p2[0];
  float distance = sqrt(((x2-x1) * (x2-x1))+((y2-y1) * (y2-y1)));
  
  return distance;

}

void displayError() {
  printf("Usage:\n");
  printf("-e edgeWidth oldImageFile newImageFile\n");
  printf("-c circleCenterRow circleCenterCol radius oldImageFile newImageFile\n");
  printf("-l p1row p1col p2row p2col oldImageFile newImageFile");
  printf("You have to run the command with the synopsis: \n\n");

  printf("./programName -e edgeWidth originalImage newImage\n");
  printf("to paint an edge of width edgeWidth in the image of originalIamge\n\n");

  printf("./programName -c circleCenterRow circleCenterCol radius oldImageFile newImageFile\n");
  printf("to paint a big round dot on the image with center at (circleCenterRow,\n");
  printf("circleCenterCol) and radius of radius\n\n");

  printf("./programName -l p1row p1col p2row p2col oldImageFile newImageFile\n");
  printf("to draw a line at a start point with row number = p1row and column\n");
  printf("number = p1col, the line segment ends at a point with row number =\n");
  printf("p2row and column number = p2co\n\n");
}