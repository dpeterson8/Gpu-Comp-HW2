#include <stdio.h>
#include <stdlib.h>

#include "pgmUtility.h"

int main(int argc, char *argv[]) {
    int i;
    FILE *in_temp = fopen("balloons.ascii.pgm", "r"); 
    FILE *out_temp = fopen("balloons.ascii-test.pgm", "w"); 
    char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));
    for(i = 0; i < rowsInHeader; i ++) {
        header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
    }

    int numRows, numCols;
    int * temp = pgmRead(header, &numRows, &numCols, in_temp);

    // for(int x = 0; x < 30; x++) {
    //     printf("%d ", temp[x]);
    // }
    int num_bytes = numCols * numRows * sizeof(int);
    int * d_temp = 0;

    hipMalloc((void **) &d_temp, num_bytes);
    hipMemcpy( d_temp, temp, num_bytes, hipMemcpyHostToDevice );
    pgmDrawCircle(d_temp, numRows, numCols, 0, 0, 10, header);
    hipDeviceSynchronize();
    hipMemcpy( temp, d_temp, num_bytes, hipMemcpyDeviceToHost );

    int awnser = pgmWrite((const char **) header, temp, numRows, numCols, out_temp);

    return 0;
}
