#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "pgmUtility.h"

int main(int argc, char *argv[]) {
  FILE *inFile;
  FILE *outFile;
  char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));

  int circleCenterRow, circleCenterCol, circleRadius;
  char originalFileName[100], newFileName[100];
  int numRows, numCols;
  int * hPixels, * dPixels;
  int num_bytes;
  char * drawType;

  if (argv[1] != NULL) {
    drawType = argv[1];
  } else { displayError(); }

  int i;
  for(i = 0; i < rowsInHeader; i ++) {
    header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
  }

  if(drawType[1] == 'c') {
    if(argc == 7) {
      circleCenterRow = atoi(argv[2]);
      circleCenterCol = atoi(argv[3]);
      circleRadius = atoi(argv[4]);
      strcpy(originalFileName, argv[5]);
      strcpy(newFileName, argv[6]);

      inFile = fopen(originalFileName, "r");
      outFile = fopen(newFileName, "w"); 

      hPixels = pgmRead(header, &numRows, &numCols, inFile);
      num_bytes = numCols * numRows * sizeof(int);

      cpuPgmDrawCircle(hPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);

      hipMalloc((void **) &dPixels, num_bytes);
      hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
      pgmDrawCircle(dPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);
      hipDeviceSynchronize();
      hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
      hipFree(dPixels);

      int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
      for(i = 0; i < rowsInHeader; i++) {
          free(header[i]);
      }
      free(header);
      free(hPixels);
    } else {
      displayError();
    }      

  } else if(drawType[1] == 'e') {

  } else if(drawType[1] == 'l') {
        
  } else { displayError(); }

  return 0;
}
