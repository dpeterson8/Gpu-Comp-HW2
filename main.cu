#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "pgmUtility.h"
#include "pgmCpuUtility.h"


int main(int argc, char *argv[]) {
  FILE *inFile;
  FILE *outFile;
  char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));

  int circleCenterRow, circleCenterCol, circleRadius;
  int p1row, p1col, p2row, p2col;
  char originalFileName[100], newFileName[100];
  int numRows, numCols;
  int * hPixels, * dPixels;
  int num_bytes;
  char * drawType;
  double now, then;
  double scost;

  if (argv[1] != NULL) {
    drawType = argv[1];
  } else { displayError(); }

  int i;
  for(i = 0; i < rowsInHeader; i ++) {
    header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
  }

  if(drawType[1] == 'c') {
    if(argc != 7) {
      displayError();
    } 
      
    // get circle col, row, and radius from argv
    circleCenterRow = atoi(argv[2]);
    circleCenterCol = atoi(argv[3]);
    circleRadius = atoi(argv[4]);

    
    strcpy(originalFileName, argv[5]);
    strcpy(newFileName, argv[6]);

    inFile = fopen(originalFileName, "r");
    outFile = fopen(newFileName, "w"); 

    if(inFile == NULL) {
      return 0;
    }

    // read file and get the header, number of rows, and number of columns in pgm file
    hPixels = pgmRead(header, &numRows, &numCols, inFile);
    num_bytes = numCols * numRows * sizeof(int);

    // run cpPgmDrawCircle which will draw the circle using only cpu
    then = currentTime();
    cpuPgmDrawCircle(hPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);
    now = currentTime();
    scost = now - then;
    printf("Drawing the circle using cpu took: %lf\n", scost);

    // calls to draw circle using 
    then = currentTime();
    hipMalloc((void **) &dPixels, num_bytes);
    hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
    pgmDrawCircle(dPixels, numRows, numCols, circleCenterCol, circleCenterRow, circleRadius, header);
    hipDeviceSynchronize();
    hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
    hipFree(dPixels);
    now = currentTime();
    scost = now - then;
    printf("Allocating memory and drawing the circle using gpu took: %lf\n", scost);

    int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
    for(i = 0; i < rowsInHeader; i++) {
        free(header[i]);
    }
    free(header); 
    free(hPixels);


  } 
  else if(drawType[1] == 'e') {

    if(argc != 5) {
      displayError();
      exit(0);
    }

    int edgeWidth = atoi(argv[2]);
    strcpy(originalFileName, argv[3]);
    strcpy(newFileName, argv[4]);

    inFile = fopen(originalFileName, "r");
    outFile = fopen(newFileName, "w"); 
    cpuPgmDrawEdge(hPixels, numRows, numCols, edgeWidth, header);
    hPixels = pgmRead(header, &numRows, &numCols, inFile);
    num_bytes = numCols * numRows * sizeof(int);

    then = currentTime();

    now = currentTime();
    scost = now - then;
    printf("Drawing the edge using cpu took: %lf\n", scost);

    then = currentTime();
    hipMalloc((void **) &dPixels, num_bytes);
    hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
    pgmDrawEdge(dPixels, numRows, numCols, edgeWidth, header);
    hipDeviceSynchronize();
    hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
    hipFree(dPixels);
    now = currentTime();
    scost = now - then;
    printf("Allocating memory and drawing the edge using gpu took: %lf\n", scost);

    int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
    for(i = 0; i < rowsInHeader; i++) {
        free(header[i]);
    }
    free(header);
    free(hPixels);

  } 
  else if(drawType[1] == 'l') {

    p1row = atoi(argv[2]);
    p1col = atoi(argv[3]);
    p2row = atoi(argv[4]);
    p2col = atoi(argv[5]);
    
    strcpy(originalFileName, argv[6]);
    strcpy(newFileName, argv[7]);

    inFile = fopen(originalFileName, "r");
    outFile = fopen(newFileName, "w"); 

    hPixels = pgmRead(header, &numRows, &numCols, inFile);
    num_bytes = numCols * numRows * sizeof(int);

    hipMalloc((void **) &dPixels, num_bytes);
    hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
    pgmDrawLine(dPixels, numRows, numCols, header, p1row, p1col, p2row, p2col);
    hipDeviceSynchronize();
    hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
    hipFree(dPixels);

    int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
    for(i = 0; i < rowsInHeader; i++) {
        free(header[i]);
    }
    free(header);
    free(hPixels);

  } else { displayError(); }

  return 0;
}
