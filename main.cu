#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "pgmUtility.h"

int main(int argc, char *argv[]) {
  FILE *inFile;
  FILE *outFile;
  char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));

  int circleCenterRow, circleCenterCol, circleRadius;
  int p1row, p1col, p2row, p2col;
  char originalFileName[100], newFileName[100];
  int numRows, numCols;
  int * hPixels, * dPixels;
  int num_bytes;
  char * drawType;

  if (argv[1] != NULL) {
    drawType = argv[1];
  } else { displayError(); }

  int i;
  for(i = 0; i < rowsInHeader; i ++) {
    header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
  }

  if(drawType[1] == 'c') {
    if(argc == 7) {
      circleCenterRow = atoi(argv[2]);
      circleCenterCol = atoi(argv[3]);
      circleRadius = atoi(argv[4]);
      strcpy(originalFileName, argv[5]);
      strcpy(newFileName, argv[6]);

      inFile = fopen(originalFileName, "r");
      outFile = fopen(newFileName, "w"); 

      hPixels = pgmRead(header, &numRows, &numCols, inFile);
      num_bytes = numCols * numRows * sizeof(int);

      cpuPgmDrawCircle(hPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);

      hipMalloc((void **) &dPixels, num_bytes);
      hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
      pgmDrawCircle(dPixels, numRows, numCols, circleCenterCol, circleCenterRow, circleRadius, header);
      hipDeviceSynchronize();
      hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
      hipFree(dPixels);

      int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
      for(i = 0; i < rowsInHeader; i++) {
          free(header[i]);
      }
      free(header);
      free(hPixels);
    } else {
      displayError();
    }      

  } else if(drawType[1] == 'e') {
      int edgeWidth = atoi(argv[2]);
      strcpy(originalFileName, argv[3]);
      strcpy(newFileName, argv[4]);

      inFile = fopen(originalFileName, "r");
      outFile = fopen(newFileName, "w"); 

      hPixels = pgmRead(header, &numRows, &numCols, inFile);
      num_bytes = numCols * numRows * sizeof(int);

      hipMalloc((void **) &dPixels, num_bytes);
      hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
      pgmDrawEdge(dPixels, numRows, numCols, edgeWidth, header);
      hipDeviceSynchronize();
      hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
      hipFree(dPixels);

      int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
      for(i = 0; i < rowsInHeader; i++) {
          free(header[i]);
      }
      free(header);
      free(hPixels);
  } else if(drawType[1] == 'l') {
      p1row = atoi(argv[2]);
      p1col = atoi(argv[3]);
      p2row = atoi(argv[4]);
      p2col = atoi(argv[5]);
      strcpy(originalFileName, argv[6]);
      strcpy(newFileName, argv[7]);

      inFile = fopen(originalFileName, "r");
      outFile = fopen(newFileName, "w"); 

      hPixels = pgmRead(header, &numRows, &numCols, inFile);
      num_bytes = numCols * numRows * sizeof(int);

      hipMalloc((void **) &dPixels, num_bytes);
      hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
      pgmDrawLine(dPixels, numRows, numCols, header, p1row, p1col, p2row, p2col);
      hipDeviceSynchronize();
      hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
      hipFree(dPixels);

      int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
      for(i = 0; i < rowsInHeader; i++) {
          free(header[i]);
      }
      free(header);
      free(hPixels);
  } else { displayError(); }

  return 0;
}
