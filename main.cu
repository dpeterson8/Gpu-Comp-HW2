#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "pgmUtility.h"
#include "pgmCpuUtility.h"


int main(int argc, char *argv[]) {
  FILE *inFile;
  FILE *outFile;
  char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));

  int circleCenterRow, circleCenterCol, circleRadius;
  int p1row, p1col, p2row, p2col;
  char originalFileName[100], newFileName[100];
  int numRows, numCols;
  int * hPixels, * dPixels;
  int num_bytes;
  char * drawType;
  // double now, then;
  // double scost, pcost;

  if (argv[1] != NULL) {
    drawType = argv[1];
  } else {
    displayError();
    exit(1);
  }

  int i;
  for(i = 0; i < rowsInHeader; i ++) {
    header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
  }

  if(drawType[1] == 'c') {
    if(argc != 7) {
      displayError();
      exit(1);
    } 
      
    // get circle col, row, and radius from argv
    circleCenterRow = atoi(argv[2]);
    circleCenterCol = atoi(argv[3]);
    circleRadius = atoi(argv[4]);
    
    strcpy(originalFileName, argv[5]);
    strcpy(newFileName, argv[6]);

    if(fopen(originalFileName, "r") != NULL) {
      inFile = fopen(originalFileName, "r");
    } else {
      displayError();
      exit(1);
    }
    outFile = fopen(newFileName, "w"); 

    if(inFile == NULL) {
      return 0;
    }

    // read file and get the header, number of rows, and number of columns in pgm file
    hPixels = pgmRead(header, &numRows, &numCols, inFile);
    fclose(inFile);
    num_bytes = numCols * numRows * sizeof(int);

    // run cpPgmDrawCircle which will draw the circle using only cpu
    // cpuPgmDrawCircle(hPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);

    // calls to draw circle using 
    hipMalloc((void **) &dPixels, num_bytes);
    hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
    pgmDrawCircle(dPixels, numRows, numCols, circleCenterCol, circleCenterRow, circleRadius, header);
    hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
    hipFree(dPixels);

    int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
    fclose(outFile);
    for(i = 0; i < rowsInHeader; i++) {
        free(header[i]);
    }
    free(header); 
    free(hPixels);


  } 
  else if(drawType[1] == 'e') {

    if(argc != 5) {
      displayError();
      exit(1);
    }

    int edgeWidth = atoi(argv[2]);
    strcpy(originalFileName, argv[3]);
    strcpy(newFileName, argv[4]);

    if(fopen(originalFileName, "r") != NULL) {
      inFile = fopen(originalFileName, "r");
    } else {
      displayError();
      exit(1);
    }

    outFile = fopen(newFileName, "w");

    hPixels = pgmRead(header, &numRows, &numCols, inFile);
    fclose(inFile);
    num_bytes = numCols * numRows * sizeof(int);

    hipMalloc((void **) &dPixels, num_bytes);
    hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
    pgmDrawEdge(dPixels, numRows, numCols, edgeWidth, header);
    hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
    hipFree(dPixels);

    int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
    fclose(outFile);
    for(i = 0; i < rowsInHeader; i++) {
        free(header[i]);
    }
    free(header);
    free(hPixels);

  } 
  else if(drawType[1] == 'l') {

    if(argc != 8) {
      displayError();
      exit(0);
    }

    p1row = atoi(argv[2]);
    p1col = atoi(argv[3]);
    p2row = atoi(argv[4]);
    p2col = atoi(argv[5]);
    
    strcpy(originalFileName, argv[6]);
    strcpy(newFileName, argv[7]);

    if(fopen(originalFileName, "r") != NULL) {
      inFile = fopen(originalFileName, "r");
    } else {
      displayError();
      exit(1);
    }
    outFile = fopen(newFileName, "w"); 

    hPixels = pgmRead(header, &numRows, &numCols, inFile);
    fclose(inFile);
    num_bytes = numCols * numRows * sizeof(int);

    hipMalloc((void **) &dPixels, num_bytes);
    hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
    pgmDrawLine(dPixels, numRows, numCols, header, p1row, p1col, p2row, p2col);
    hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
    hipFree(dPixels);

    int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
    fclose(outFile);
    for(i = 0; i < rowsInHeader; i++) {
        free(header[i]);
    }
    free(header);
    free(hPixels);

  } else { 
    displayError();
    exit(1);
  }

  return 0;
}
