#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "pgmUtility.h"

int main(int argc, char *argv[]) {
    FILE *inFile;
    FILE *outFile;
    char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));

    int circleCenterRow, circleCenterCol, circleRadius;
    char originalFileName[100], newFileName[100];
    int numRows, numCols;
    int * hPixels, * dPixels;
    int num_bytes;

    char * drawType = argv[1];

    int i;
    for(i = 0; i < rowsInHeader; i ++) {
        header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
    }

    if(drawType[1] == 'c') {
        
        circleCenterRow = atoi(argv[2]);
        circleCenterCol = atoi(argv[3]);
        circleRadius = atoi(argv[4]);
        strcpy(originalFileName, argv[5]);
        strcpy(newFileName, argv[6]);

        inFile = fopen(originalFileName, "r");
        outFile = fopen("balloons.ascii-test.pgm", "w"); 

        hPixels = pgmRead(header, &numRows, &numCols, inFile);
        num_bytes = numCols * numRows * sizeof(int);

        cpuPgmDrawCircle(hPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);

        hipMalloc((void **) &dPixels, num_bytes);
        hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
        pgmDrawCircle(dPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);
        hipDeviceSynchronize();
        hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
        hipFree(dPixels);

        int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
        for(i = 0; i < rowsInHeader; i++) {
            free(header[i]);
        }
        free(header);
        free(hPixels);

    } else if(drawType[1] == 'e') {

    } else if(drawType[1] == 'l') {
        
    }

    return 0;
}
