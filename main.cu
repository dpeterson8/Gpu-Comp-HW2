#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "pgmUtility.h"


int main(int argc, char *argv[]) {
  FILE *inFile;
  FILE *outFile;
  char **header = ( char** ) malloc(rowsInHeader * sizeof(char *));

  int circleCenterRow, circleCenterCol, circleRadius;
  char originalFileName[100], newFileName[100];
  int numRows, numCols;
  int * hPixels, * dPixels;
  int num_bytes;
  char * drawType;

  if (argv[1] != NULL) {
    drawType = argv[1];
  } else { displayError(); }

  int i;
  for(i = 0; i < rowsInHeader; i ++) {
    header[i] = (char* ) malloc(sizeof(char) * maxSizeHeadRow);
  }

  if(drawType[1] == 'c') {
    if(argc == 7) {
      circleCenterRow = atoi(argv[2]);
      circleCenterCol = atoi(argv[3]);
      circleRadius = atoi(argv[4]);
      strcpy(originalFileName, argv[5]);
      strcpy(newFileName, argv[6]);

      inFile = fopen(originalFileName, "r");
      outFile = fopen(newFileName, "w"); 

      hPixels = pgmRead(header, &numRows, &numCols, inFile);
      num_bytes = numCols * numRows * sizeof(int);

      cpuPgmDrawCircle(hPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);

      hipMalloc((void **) &dPixels, num_bytes);
      hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
      pgmDrawCircle(dPixels, numRows, numCols, circleCenterRow, circleCenterCol, circleRadius, header);
      hipDeviceSynchronize();
      hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
      hipFree(dPixels);

      int ret = pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
      for(i = 0; i < rowsInHeader; i++) {
          free(header[i]);
      }
      free(header);
      free(hPixels);
    } else {
      displayError();
    }      

    } else if(drawType[1] == 'e') {
        
        

    } else if(drawType[1] == 'l') {
        printf("Entering line draw mode\n");
        //TODO: Error checking 
        //Parse Parameters
        int p1x, p1y, p2x, p2y;
        p1x = atoi(argv[2]); 
        p1y = atoi(argv[3]);
        p2x = atoi(argv[4]);
        p2y = atoi(argv[5]);

        strcpy(originalFileName, argv[6]);
        strcpy(newFileName, argv[7]);
        //Open file streams
        inFile = fopen(originalFileName, "r");
        outFile = fopen(newFileName, "w");
        //Time and launch CPU code
        //cpuPgmDrawLine(hPixels, numRows, numCols, header, p1x, p1y, p2x, p2y);

        //Time and launch Kernel

        //cuda Memcpy
        hPixels = pgmRead(header, &numRows, &numCols, inFile);
        num_bytes = numCols * numRows * sizeof(int);

        hipMalloc((void **) &dPixels, num_bytes);
        hipMemcpy( dPixels, hPixels, num_bytes, hipMemcpyHostToDevice );
        pgmDrawLine(hPixels, numRows, numCols, header, p1x, p1y, p2x, p2y);
        hipDeviceSynchronize();
        hipMemcpy( hPixels, dPixels, num_bytes, hipMemcpyDeviceToHost );
        hipFree(dPixels);
        //Print Time

        //Write to new pgm with one of these results
        pgmWrite((const char **) header, hPixels, numRows, numCols, outFile);
        printf("Should've made pgm image\n");
        //Free memory
        for(i = 0; i < rowsInHeader; i++) {
            free(header[i]);
        }
        free(header);
        free(hPixels);
    }

  return 0;
}
